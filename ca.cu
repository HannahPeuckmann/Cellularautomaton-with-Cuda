#include "hip/hip_runtime.h"
/* Cellularautomaton, paralell implementation with CUDA
 * Hannah Peuckmann, Matr.-Nr.:791996, WiSe 2021/22
 * #1: Number of lines
 * #2: Number of iterations to be simulated
 *
 */
#include <cstdio>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <assert.h>
#include "openssl/md5.h"
#include "random.h"
#include <fcntl.h>


/* horizontal size of the configuration */
#define XSIZE 1024

/* "ADT" State and line of states (plus border) */
typedef char State;
typedef State Line[XSIZE + 2];

// Error checking and time measurement setup
#define CUDA_ERR_CHECK(x) do { hipError_t err = x; if (( err ) != hipSuccess ) { \
  printf ("Error \"%s\" at %s :%d \n" , hipGetErrorString(err), \
        __FILE__ , __LINE__ ) ; exit(-1); \
}} while (0)

#define TIME_GET(timer) \
  struct timespec timer; \
  clock_gettime(CLOCK_MONOTONIC, &timer)

#define TIME_DIFF(timer1, timer2) \
  ((timer2.tv_sec * 1.0E+9 + timer2.tv_nsec) - \
  (timer1.tv_sec * 1.0E+9 + timer1.tv_nsec)) / 1.0E+9

/* determine random integer between 0 and n-1 */
#define randInt(n) ((int)(nextRandomLEcuyer() * n))

/* get MD5 checksum string of a memory chunk */
char* getMD5DigestStr(void* buf, size_t buflen)
{
  MD5_CTX ctx;
	unsigned char sum[MD5_DIGEST_LENGTH];
	int i;
	char* retval;
	char* ptr;

	MD5_Init(&ctx);
	MD5_Update(&ctx, buf, buflen);
	MD5_Final(sum, &ctx);

	retval = (char*)calloc(MD5_DIGEST_LENGTH * 2 + 1, sizeof(*retval));
	ptr = retval;

	for (i = 0; i < MD5_DIGEST_LENGTH; i++) {
		snprintf(ptr, 3, "%02X", sum[i]);
		ptr += 2;
	}

	return retval;
}


/* --------------------- CA simulation -------------------------------- */

/* random starting configuration */
static void initConfig(Line *buf, int lines)
{
	int x, y;

	initRandomLEcuyer(424243);
	for (y = 1;  y <= lines;  y++) {
		for (x = 1;  x <= XSIZE;  x++) {
			buf[y][x] = randInt(100) >= 50;
		}
	}
}


/* treat torus like boundary conditions */
__global__ static void boundary(Line *buf, int lines)
{
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;

  if (y == lines-1) {
    buf[y+2][x+1] = buf[1][x+1];
  }
  if (y == 0) {
    buf[y][x+1] = buf[lines][x+1];
  }
  // We only have 'lines' times threads on the y axis so the edges are not copied over
  if (x == 0) {
    buf[y+1][x] = buf[y+1][XSIZE];
  }
  if (x == XSIZE-1) {
    buf[y+1][x+2] = buf[y+1][1];
  }

  // Extra step to copy over the four edges, make sure all the steps above are finished to ensure that there is data to copy from
  if (x == 0 && y == 0){
    buf[0][0] = buf[lines][XSIZE];
    buf[0][XSIZE+1] = buf[lines][1];
    buf[lines+1][0] = buf[1][XSIZE];
    buf[lines+1][XSIZE+1] = buf[1][1];

  }

}

/* make one simulation iteration with 'lines' lines.
* old configuration is in from, new one is written to 'to'.
*/
__global__ void simulate(Line *from, Line *to, int lines)
{

  static State anneal[10] = {0, 0, 0, 0, 1, 0, 1, 1, 1, 1};
  int y = (blockIdx.y * blockDim.y + threadIdx.y) +1;
  int x = (blockIdx.x * blockDim.x + threadIdx.x) +1;
  to[y][x]= anneal[(from)[(y)-1][(x)-1] + (from)[(y)][(x)-1] + (from)[(y)+1][(x)-1] +\
          (from)[(y)-1][(x)  ] + (from)[(y)][(x)  ] + (from)[(y)+1][(x)  ] +\
          (from)[(y)-1][(x)+1] + (from)[(y)][(x)+1] + (from)[(y)+1][(x)+1]];
}


/* --------------------- measurement ---------------------------------- */

int main(int argc, char** argv)
{
  int lines, iterations;
  Line *pFrom, *pTo, *pTemp; // device
  char* hash;

  assert(argc == 3);

  lines = atoi(argv[1]);
  iterations = atoi(argv[2]);

  from = (Line*)calloc((lines + 2), sizeof(Line));
  to   = (Line*)calloc((lines + 2), sizeof(Line));

  initConfig(from, lines);

  if(from == NULL | to == NULL){
    printf("ERROR: failure allocating host memory\n");
    exit(EXIT_FAILURE);
  }

  // allocate device memory
  hipMalloc((void **)&pFrom, (lines +2) * sizeof(Line));
  hipMalloc((void **)&pTo, (lines +2) * sizeof(Line));

  if(pFrom == NULL | pTo == NULL){
    printf("ERROR: failure allocating device memory\n");
    exit(EXIT_FAILURE);
  }

  TIME_GET(start);
  dim3 numBlocks(32, 4096);
  dim3 threadsPerBlock(32, 32); // max threads per block = 1024

  hipMemcpy(pFrom, from, (lines +2) * sizeof(Line), hipMemcpyHostToDevice);
  for (int i= 0; i < iterations; i++) {
    boundary<<<numBlocks, threadsPerBlock>>>(pFrom, lines);
    simulate<<<numBlocks, threadsPerBlock>>>(pFrom, pTo, lines);
    pTemp = pFrom;
    pFrom = pTo;
    pTo = pTemp;
  }
  hipMemcpy(from, pFrom, (lines+2) * sizeof(Line), hipMemcpyDeviceToHost);
  CUDA_ERR_CHECK( hipGetLastError() );
  TIME_GET(end);

	hash = getMD5DigestStr(from,lines);
	printf("hash gpu: %s\ttime: %.1f ms\n", hash, TIME_DIFF(start,end)*1000);

  hipFree(pFrom);
  hipFree(pTo);
  hipFree(pTemp);

  free(from);
  free(to);
  free(hash);
  return EXIT_SUCCESS;
}